#include "hip/hip_runtime.h"
#include <cstdio> 
#include <cstdlib> // provides malloc, rand, exit
#include <iostream> // for printing to the terminal std::cout and std::cerr
#include <hip/hip_runtime.h> // for cuda runtime api functions like cudamalloc, cudamemcpy

__global__ void vector_add(int *a, int *b, int *c,int N) {

    int i= blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N){
        c[i] = a[i] + b[i];
    }
}

__global__ void initialize_data(int *a, int *b, int N){
    int i= blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i<N){
        a[i] = i;
        b[i] = 2*i;
    }

}

int main () {
    
    int N,
    int threads=32;
    int blocks= (N+threads-1)/N;

    // host memory ptr = (type*)malloc(size); free(ptr);
    int *ha = (int*)malloc(N*sizeof(int)); // int* means using malloc[memory allocation] for integer storage, pointer ha has the address of memory allocated.
    int *hb = (int*)malloc(N*sizeof(int));
    int *hc = (int*)malloc(N*sizeof(int));
     
    //device memory hipMalloc((void**)&d_ptr, size); hipFree(d_ptr);
    int *da, *db, *dc;
    hipMalloc((void**)&da,N*sizeof(int));
    hipMalloc((void**)&db,N*sizeof(int));
    hipMalloc((void**)&dc,N*sizeof(int));

    // copy from host to device
    hipMemcpy(da,ha,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(db,hb,N*sizeof(int),hipMemcpyHostToDevice);

    // intitalize data and perform additon
    initialize_data<<<blocks,threads>>>(da,db,N);
    vector_add<<<blocks,threads>>>(da,db,dc,N);
    hipDeviceSynchronize();
    
    // copy result back to host
    hipMemcpy(hc,dc,N*sizeof(int),hipMemcpyDeviceToHost);

    bool success=true;
    for (i=0:i<N:++i){
        if(hc[i]!= ha[i]+hb[i]){
                std::cerr << "mismatch at index" << i <<":" <<"hc[i]"<<"!=" <<ha[i]+hb[i] <<std::cendl;
                success=false;
                break;
        }
    }

    std::cout<< (success? "vector additon successful ": "failed operation") << std::endl;
    free(ha);
    free(hb);
    free(hc);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}


/*
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

*/