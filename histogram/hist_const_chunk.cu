#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstdint>


__constant__ uint32_t const_mem[16384];

__global__ void hist(uint32_t *bins_d, int chunk_size, int num_bins){
    int i = blockIdx.x * blockDim.x + threadIdx.x; // row
    if(i < chunk_size){
        uint32_t val = const_mem[i];
        atomicAdd(&bins_d[val],1);
    }
}



int main(){
    int chunk_size=16384;
    int num_bins=4096;
    int len=65536;
    uint32_t *input=(uint32_t*)malloc(len*sizeof(uint32_t));
    uint32_t *bins=(uint32_t*)malloc(num_bins*sizeof(uint32_t));
    uint32_t *bins_d;

    //hipMalloc((void**)&input_d,len*sizeof(uint32_t)); // cuda malloc is generic allocator &input_d is uint32_t** type, casting to void**
    hipMalloc((void**)&bins_d,num_bins*sizeof(uint32_t));   // for global memory

    for (int i=0;i<len;++i){
        input[i]=rand()% num_bins;
    }

    for (int i = 0; i < num_bins; ++i) // here i++,++i work same because result of expression isn't used.
    {
        bins[i]=0;
    }

    hipMemcpy(bins_d,bins,num_bins*sizeof(uint32_t),hipMemcpyHostToDevice);
  
    for(int i=0;i<len;i+=chunk_size){
        int current_chunk=min(chunk_size,len-i);
        hipMemcpyToSymbol(HIP_SYMBOL(const_mem),input+i,current_chunk*sizeof(uint32_t));
        int threads= 256;
        int blocks=(current_chunk+threads-1)/threads;

        hist<<<blocks,threads>>>(bins_d,chunk_size,num_bins);
        hipDeviceSynchronize();
    }


    hipMemcpy(bins,bins_d,num_bins*sizeof(uint32_t),hipMemcpyDeviceToHost);



    for(int i=0;i<100;++i){
        printf("bins[%d]:%u\n",i,bins[i]);
    }

    hipFree(bins_d);
    hipFree(const_mem);
    free(bins);
    free(input);

    return 0;
  
}